/**
* \file		Cub.cu
* \author	Daniel Meister
* \date		2017/01/23
* \brief	Cub wrapper source file.
*/

#include "Cub.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

namespace Cub {

template <typename T>
float sort(
	int numberOfItems,
	T * keys0,
	T * keys1,
	int * values0,
	int * values1,
	bool & swapBuffers
	) {

	hipcub::DoubleBuffer<T> keysBuffer(keys0, keys1);
	hipcub::DoubleBuffer<int> valuesBuffer(values0, values1);

	void * tempStorage = nullptr;
	size_t storageSize = 0;
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer, numberOfItems);
	hipMalloc(&tempStorage, storageSize);

	float elapsedTime = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	hipcub::DeviceRadixSort::SortPairs(tempStorage, storageSize, keysBuffer, valuesBuffer, numberOfItems);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipFree(tempStorage);

	swapBuffers = keysBuffer.selector != 0;

	return elapsedTime * 1.0e-3f;

}

float sort(
	int numberOfItems,
	unsigned int * keys0,
	unsigned int * keys1,
	int * values0,
	int * values1,
	bool & swapBuffers
	) {
	return sort<unsigned int>(numberOfItems, keys0, keys1, values0, values1, swapBuffers);
}

float sort(
	int numberOfItems,
	unsigned long long * keys0,
	unsigned long long * keys1,
	int * values0,
	int * values1,
	bool & swapBuffers
	) {
	return sort<unsigned long long int>(numberOfItems, keys0, keys1, values0, values1, swapBuffers);
}

};